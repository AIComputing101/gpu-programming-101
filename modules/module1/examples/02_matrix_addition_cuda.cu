#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void matrixAdd(float *A, float *B, float *C, int width, int height) {
    // 2D thread indexing
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Boundary check
    if (row < height && col < width) {
        int index = row * width + col;
        C[index] = A[index] + B[index];
    }
}

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

int main() {
    const int width = 1024;
    const int height = 1024;
    const int size = width * height * sizeof(float);
    
    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    
    // Initialize matrices
    for (int i = 0; i < width * height; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }
    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));
    
    // Copy to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));
    
    // Define block and grid sizes
    dim3 blockSize(16, 16);  // 256 threads per block
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);
    
    printf("Matrix dimensions: %d x %d\n", width, height);
    printf("Grid size: (%d, %d)\n", gridSize.x, gridSize.y);
    printf("Block size: (%d, %d)\n", blockSize.x, blockSize.y);
    printf("Total threads: %d\n", gridSize.x * gridSize.y * blockSize.x * blockSize.y);
    
    // Launch kernel
    matrixAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, width, height);
    
    // Check for errors
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy result back
    CUDA_CHECK(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
    
    // Verify result
    bool success = true;
    for (int i = 0; i < width * height; i++) {
        if (h_C[i] != 3.0f) {
            success = false;
            printf("Error at element %d: expected 3.0, got %f\n", i, h_C[i]);
            break;
        }
    }
    
    printf("Matrix addition %s\n", success ? "PASSED" : "FAILED");
    
    // Cleanup
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    
    return 0;
}