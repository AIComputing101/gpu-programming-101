#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <omp.h>

#define TRANSFER_SIZE (64 * 1024 * 1024)  // 64 MB
#define NUM_ITERATIONS 10

// Simple kernel for data processing
__global__ void processData(float *data, int n, int gpu_id) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        // Simple computation with GPU identifier
        data[idx] = data[idx] * (gpu_id + 1.0f) + sinf(idx * 0.001f);
    }
}

// Kernel to verify data integrity
__global__ void verifyData(float *data, float *expected, bool *result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float diff = fabsf(data[idx] - expected[idx]);
        if (diff > 1e-5f) {
            *result = false;
        }
    }
}

// Simple addition kernel for peer-to-peer communication
__global__ void addArrays(float *result, float *input, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        result[idx] += input[idx];
    }
}

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

void checkP2PCapabilities() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    
    printf("Peer-to-Peer Capabilities Analysis:\n");
    printf("===================================\n");
    printf("Number of GPUs: %d\n\n", deviceCount);
    
    if (deviceCount < 2) {
        printf("P2P communication requires at least 2 GPUs.\n");
        return;
    }
    
    // Check P2P access capabilities
    printf("P2P Access Matrix:\n");
    printf("From\\To  ");
    for (int j = 0; j < deviceCount; j++) {
        printf("GPU%d ", j);
    }
    printf("\n");
    
    for (int i = 0; i < deviceCount; i++) {
        printf("GPU%d     ", i);
        for (int j = 0; j < deviceCount; j++) {
            if (i == j) {
                printf("---- ");
            } else {
                int canAccessPeer;
                CUDA_CHECK(hipDeviceCanAccessPeer(&canAccessPeer, i, j));
                printf(" %s  ", canAccessPeer ? "Yes" : "No");
            }
        }
        printf("\n");
    }
    
    // Check detailed P2P attributes
    printf("\nDetailed P2P Attributes:\n");
    for (int i = 0; i < deviceCount; i++) {
        for (int j = 0; j < deviceCount; j++) {
            if (i != j) {
                printf("GPU %d to GPU %d:\n", i, j);
                
                int attr;
                CUDA_CHECK(hipDeviceGetP2PAttribute(&attr, hipDevP2PAttrPerformanceRank, i, j));
                printf("  Performance Rank: %d\n", attr);
                
                CUDA_CHECK(hipDeviceGetP2PAttribute(&attr, hipDevP2PAttrAccessSupported, i, j));
                printf("  Access Supported: %s\n", attr ? "Yes" : "No");
                
                CUDA_CHECK(hipDeviceGetP2PAttribute(&attr, hipDevP2PAttrNativeAtomicSupported, i, j));
                printf("  Native Atomics: %s\n", attr ? "Yes" : "No");
                
                CUDA_CHECK(hipDeviceGetP2PAttribute(&attr, hipDevP2PAttrHipArrayAccessSupported, i, j));
                printf("  CUDA Array Access: %s\n", attr ? "Yes" : "No");
                
                printf("\n");
            }
        }
    }
}

void enableP2PAccess(int deviceCount) {
    printf("Enabling P2P access between all capable GPU pairs...\n");
    
    for (int i = 0; i < deviceCount; i++) {
        CUDA_CHECK(hipSetDevice(i));
        for (int j = 0; j < deviceCount; j++) {
            if (i != j) {
                int canAccessPeer;
                CUDA_CHECK(hipDeviceCanAccessPeer(&canAccessPeer, i, j));
                
                if (canAccessPeer) {
                    hipError_t result = hipDeviceEnablePeerAccess(j, 0);
                    if (result == hipSuccess) {
                        printf("  GPU %d -> GPU %d: P2P enabled\n", i, j);
                    } else if (result == hipErrorPeerAccessAlreadyEnabled) {
                        printf("  GPU %d -> GPU %d: P2P already enabled\n", i, j);
                    } else {
                        printf("  GPU %d -> GPU %d: P2P enable failed (%s)\n", 
                               i, j, hipGetErrorString(result));
                    }
                }
            }
        }
    }
    printf("\n");
}

// Measure P2P bandwidth between two GPUs
double measureP2PBandwidth(int srcDevice, int dstDevice, size_t bytes) {
    CUDA_CHECK(hipSetDevice(srcDevice));
    
    float *src_data;
    CUDA_CHECK(hipMalloc(&src_data, bytes));
    
    CUDA_CHECK(hipSetDevice(dstDevice));
    
    float *dst_data;
    CUDA_CHECK(hipMalloc(&dst_data, bytes));
    
    // Initialize source data
    CUDA_CHECK(hipSetDevice(srcDevice));
    CUDA_CHECK(hipMemset(src_data, 1, bytes));
    
    // Warm up
    for (int i = 0; i < 3; i++) {
        CUDA_CHECK(hipMemcpyPeer(dst_data, dstDevice, src_data, srcDevice, bytes));
    }
    
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Measure bandwidth
    auto start = std::chrono::high_resolution_clock::now();
    
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        CUDA_CHECK(hipMemcpyPeer(dst_data, dstDevice, src_data, srcDevice, bytes));
    }
    
    CUDA_CHECK(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    
    double total_time = std::chrono::duration<double>(end - start).count();
    double bandwidth = (bytes * NUM_ITERATIONS) / (total_time * 1e9); // GB/s
    
    // Cleanup
    CUDA_CHECK(hipSetDevice(srcDevice));
    hipFree(src_data);
    CUDA_CHECK(hipSetDevice(dstDevice));
    hipFree(dst_data);
    
    return bandwidth;
}

// Measure host-to-device bandwidth for comparison
double measureHostDeviceBandwidth(int device, size_t bytes) {
    CUDA_CHECK(hipSetDevice(device));
    
    // Allocate host memory (pinned for better performance)
    float *host_data;
    CUDA_CHECK(hipHostAlloc(&host_data, bytes, hipHostMallocDefault));
    
    float *device_data;
    CUDA_CHECK(hipMalloc(&device_data, bytes));
    
    // Initialize host data
    for (size_t i = 0; i < bytes / sizeof(float); i++) {
        host_data[i] = (float)i;
    }
    
    // Warm up
    for (int i = 0; i < 3; i++) {
        CUDA_CHECK(hipMemcpy(device_data, host_data, bytes, hipMemcpyHostToDevice));
    }
    
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Measure bandwidth
    auto start = std::chrono::high_resolution_clock::now();
    
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        CUDA_CHECK(hipMemcpy(device_data, host_data, bytes, hipMemcpyHostToDevice));
    }
    
    CUDA_CHECK(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    
    double total_time = std::chrono::duration<double>(end - start).count();
    double bandwidth = (bytes * NUM_ITERATIONS) / (total_time * 1e9); // GB/s
    
    // Cleanup
    hipHostFree(host_data);
    hipFree(device_data);
    
    return bandwidth;
}

// Demonstrate asynchronous P2P transfers
void demonstrateAsyncP2P(int srcDevice, int dstDevice) {
    printf("Demonstrating asynchronous P2P transfers (GPU %d -> GPU %d)...\n", 
           srcDevice, dstDevice);
    
    const size_t chunkSize = TRANSFER_SIZE / 4;
    const int numChunks = 4;
    
    // Allocate memory on both devices
    CUDA_CHECK(hipSetDevice(srcDevice));
    float *src_data;
    CUDA_CHECK(hipMalloc(&src_data, TRANSFER_SIZE));
    
    CUDA_CHECK(hipSetDevice(dstDevice));
    float *dst_data;
    CUDA_CHECK(hipMalloc(&dst_data, TRANSFER_SIZE));
    
    // Create streams
    hipStream_t srcStream, dstStream;
    CUDA_CHECK(hipSetDevice(srcDevice));
    CUDA_CHECK(hipStreamCreate(&srcStream));
    CUDA_CHECK(hipSetDevice(dstDevice));
    CUDA_CHECK(hipStreamCreate(&dstStream));
    
    // Initialize source data
    CUDA_CHECK(hipSetDevice(srcDevice));
    dim3 block(256);
    dim3 grid((TRANSFER_SIZE / sizeof(float) + block.x - 1) / block.x);
    processData<<<grid, block>>>(src_data, TRANSFER_SIZE / sizeof(float), srcDevice);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // Pipeline P2P transfers with computation
    for (int chunk = 0; chunk < numChunks; chunk++) {
        size_t offset = chunk * chunkSize;
        
        // Async P2P copy
        CUDA_CHECK(hipMemcpyPeerAsync(dst_data + offset / sizeof(float), dstDevice,
                                      src_data + offset / sizeof(float), srcDevice,
                                      chunkSize, srcStream));
        
        // Process data on destination device
        CUDA_CHECK(hipSetDevice(dstDevice));
        dim3 chunkGrid((chunkSize / sizeof(float) + block.x - 1) / block.x);
        processData<<<chunkGrid, block, 0, dstStream>>>(
            dst_data + offset / sizeof(float), 
            chunkSize / sizeof(float), 
            dstDevice);
        CUDA_CHECK(hipGetLastError());
    }
    
    // Wait for all operations to complete
    CUDA_CHECK(hipSetDevice(srcDevice));
    CUDA_CHECK(hipStreamSynchronize(srcStream));
    CUDA_CHECK(hipSetDevice(dstDevice));
    CUDA_CHECK(hipStreamSynchronize(dstStream));
    
    auto end = std::chrono::high_resolution_clock::now();
    double time = std::chrono::duration<double, std::milli>(end - start).count();
    
    printf("  Async P2P pipeline completed in %.2f ms\n", time);
    printf("  Effective bandwidth: %.2f GB/s\n", 
           (TRANSFER_SIZE / (1024.0*1024.0*1024.0)) / (time / 1000.0));
    
    // Cleanup
    CUDA_CHECK(hipSetDevice(srcDevice));
    hipFree(src_data);
    hipStreamDestroy(srcStream);
    
    CUDA_CHECK(hipSetDevice(dstDevice));
    hipFree(dst_data);
    hipStreamDestroy(dstStream);
}

// Multi-GPU ring communication pattern
void demonstrateRingCommunication(int deviceCount) {
    if (deviceCount < 3) {
        printf("Ring communication requires at least 3 GPUs.\n");
        return;
    }
    
    printf("Demonstrating ring communication pattern with %d GPUs...\n", deviceCount);
    
    const size_t elementsPerGPU = TRANSFER_SIZE / (deviceCount * sizeof(float));
    const size_t bytesPerGPU = elementsPerGPU * sizeof(float);
    
    // Allocate data on each GPU
    float **gpu_data = new float*[deviceCount];
    hipStream_t *streams = new hipStream_t[deviceCount];
    
    for (int gpu = 0; gpu < deviceCount; gpu++) {
        CUDA_CHECK(hipSetDevice(gpu));
        CUDA_CHECK(hipMalloc(&gpu_data[gpu], bytesPerGPU));
        CUDA_CHECK(hipStreamCreate(&streams[gpu]));
        
        // Initialize data with GPU-specific pattern
        dim3 block(256);
        dim3 grid((elementsPerGPU + block.x - 1) / block.x);
        processData<<<grid, block>>>(gpu_data[gpu], elementsPerGPU, gpu);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }
    
    printf("Initial data distribution:\n");
    for (int gpu = 0; gpu < deviceCount; gpu++) {
        printf("  GPU %d: %.2f MB\n", gpu, bytesPerGPU / (1024.0*1024.0));
    }
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // Perform ring communication: each GPU sends to next GPU in ring
    for (int step = 0; step < deviceCount - 1; step++) {
        printf("Ring step %d: ", step + 1);
        
        #pragma omp parallel for
        for (int gpu = 0; gpu < deviceCount; gpu++) {
            int nextGPU = (gpu + 1) % deviceCount;
            
            CUDA_CHECK(hipSetDevice(gpu));
            
            // Allocate temporary buffer for received data
            float *temp_buffer;
            CUDA_CHECK(hipMalloc(&temp_buffer, bytesPerGPU));
            
            // Send data to next GPU in ring
            CUDA_CHECK(hipMemcpyPeerAsync(temp_buffer, nextGPU,
                                          gpu_data[gpu], gpu,
                                          bytesPerGPU, streams[gpu]));
            
            CUDA_CHECK(hipStreamSynchronize(streams[gpu]));
            
            // Copy received data to main buffer on destination GPU
            CUDA_CHECK(hipSetDevice(nextGPU));
            CUDA_CHECK(hipMemcpyAsync(gpu_data[nextGPU], temp_buffer,
                                      bytesPerGPU, hipMemcpyDeviceToDevice, 
                                      streams[nextGPU]));
            
            CUDA_CHECK(hipSetDevice(gpu));
            hipFree(temp_buffer);
        }
        
        // Synchronize all GPUs
        for (int gpu = 0; gpu < deviceCount; gpu++) {
            CUDA_CHECK(hipSetDevice(gpu));
            CUDA_CHECK(hipStreamSynchronize(streams[gpu]));
        }
        
        printf("completed\n");
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    double time = std::chrono::duration<double, std::milli>(end - start).count();
    
    printf("Ring communication completed in %.2f ms\n", time);
    printf("Total data transferred: %.2f MB\n", 
           (bytesPerGPU * deviceCount * (deviceCount - 1)) / (1024.0*1024.0));
    printf("Average bandwidth: %.2f GB/s\n", 
           (bytesPerGPU * deviceCount * (deviceCount - 1)) / 
           (1024.0*1024.0*1024.0) / (time / 1000.0));
    
    // Cleanup
    for (int gpu = 0; gpu < deviceCount; gpu++) {
        CUDA_CHECK(hipSetDevice(gpu));
        hipFree(gpu_data[gpu]);
        hipStreamDestroy(streams[gpu]);
    }
    
    delete[] gpu_data;
    delete[] streams;
}

// All-reduce operation using P2P
void demonstrateAllReduce(int deviceCount) {
    if (deviceCount < 2) {
        printf("All-reduce requires at least 2 GPUs.\n");
        return;
    }
    
    printf("Demonstrating all-reduce operation with %d GPUs...\n", deviceCount);
    
    const int elementsPerGPU = 1024 * 1024; // 1M floats per GPU
    const size_t bytesPerGPU = elementsPerGPU * sizeof(float);
    
    float **gpu_data = new float*[deviceCount];
    float **gpu_result = new float*[deviceCount];
    float *host_verify = new float[elementsPerGPU];
    
    // Initialize data on each GPU
    for (int gpu = 0; gpu < deviceCount; gpu++) {
        CUDA_CHECK(hipSetDevice(gpu));
        CUDA_CHECK(hipMalloc(&gpu_data[gpu], bytesPerGPU));
        CUDA_CHECK(hipMalloc(&gpu_result[gpu], bytesPerGPU));
        
        // Initialize with GPU-specific values (gpu + 1)
        float *temp_data = new float[elementsPerGPU];
        for (int i = 0; i < elementsPerGPU; i++) {
            temp_data[i] = (float)(gpu + 1);
        }
        
        CUDA_CHECK(hipMemcpy(gpu_data[gpu], temp_data, bytesPerGPU, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(gpu_result[gpu], temp_data, bytesPerGPU, hipMemcpyHostToDevice));
        
        delete[] temp_data;
    }
    
    // Calculate expected result for verification
    float expected_sum = 0.0f;
    for (int gpu = 0; gpu < deviceCount; gpu++) {
        expected_sum += (gpu + 1);
    }
    
    for (int i = 0; i < elementsPerGPU; i++) {
        host_verify[i] = expected_sum;
    }
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // Perform all-reduce: sum data from all GPUs
    for (int srcGPU = 0; srcGPU < deviceCount; srcGPU++) {
        for (int dstGPU = 0; dstGPU < deviceCount; dstGPU++) {
            if (srcGPU != dstGPU) {
                // Allocate temporary buffer on destination GPU
                CUDA_CHECK(hipSetDevice(dstGPU));
                float *temp_buffer;
                CUDA_CHECK(hipMalloc(&temp_buffer, bytesPerGPU));
                
                // Copy data from source to destination
                CUDA_CHECK(hipMemcpyPeer(temp_buffer, dstGPU,
                                         gpu_data[srcGPU], srcGPU, bytesPerGPU));
                
                // Add to result (element-wise addition)
                dim3 block(256);
                dim3 grid((elementsPerGPU + block.x - 1) / block.x);
                
                // Launch addition kernel
                addArrays<<<grid, block>>>(gpu_result[dstGPU], temp_buffer, elementsPerGPU);
                CUDA_CHECK(hipGetLastError());
                CUDA_CHECK(hipDeviceSynchronize());
                
                hipFree(temp_buffer);
            }
        }
    }
    
    auto end = std::chrono::high_resolution_clock::now();
    double time = std::chrono::duration<double, std::milli>(end - start).count();
    
    // Verify results
    printf("All-reduce completed in %.2f ms\n", time);
    
    bool all_correct = true;
    for (int gpu = 0; gpu < deviceCount; gpu++) {
        CUDA_CHECK(hipSetDevice(gpu));
        
        float *host_result = new float[elementsPerGPU];
        CUDA_CHECK(hipMemcpy(host_result, gpu_result[gpu], bytesPerGPU, 
                             hipMemcpyDeviceToHost));
        
        bool gpu_correct = true;
        for (int i = 0; i < 100; i++) { // Check first 100 elements
            if (fabsf(host_result[i] - expected_sum) > 1e-5f) {
                gpu_correct = false;
                break;
            }
        }
        
        printf("  GPU %d result verification: %s (expected %.1f, got %.1f)\n", 
               gpu, gpu_correct ? "PASS" : "FAIL", expected_sum, host_result[0]);
        
        if (!gpu_correct) all_correct = false;
        delete[] host_result;
    }
    
    printf("Overall verification: %s\n", all_correct ? "PASS" : "FAIL");
    
    // Cleanup
    for (int gpu = 0; gpu < deviceCount; gpu++) {
        CUDA_CHECK(hipSetDevice(gpu));
        hipFree(gpu_data[gpu]);
        hipFree(gpu_result[gpu]);
    }
    
    delete[] gpu_data;
    delete[] gpu_result;
    delete[] host_verify;
}

int main() {
    printf("Peer-to-Peer Communication Demonstration\n");
    printf("========================================\n\n");
    
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    
    if (deviceCount < 2) {
        printf("This demonstration requires at least 2 GPUs. Found %d GPU(s).\n", deviceCount);
        return 1;
    }
    
    // Check P2P capabilities
    checkP2PCapabilities();
    
    // Enable P2P access
    enableP2PAccess(deviceCount);
    
    // Bandwidth measurements
    printf("=== Bandwidth Measurements ===\n");
    
    const size_t testSize = TRANSFER_SIZE;
    printf("Transfer size: %.2f MB\n", testSize / (1024.0*1024.0));
    printf("Number of iterations: %d\n\n", NUM_ITERATIONS);
    
    // Measure host-device bandwidth for comparison
    printf("Host-Device Bandwidth:\n");
    for (int gpu = 0; gpu < deviceCount; gpu++) {
        double bandwidth = measureHostDeviceBandwidth(gpu, testSize);
        printf("  Host -> GPU %d: %.2f GB/s\n", gpu, bandwidth);
    }
    printf("\n");
    
    // Measure P2P bandwidth
    printf("P2P Bandwidth Matrix:\n");
    printf("From\\To  ");
    for (int j = 0; j < deviceCount; j++) {
        printf("GPU%d     ", j);
    }
    printf("\n");
    
    for (int i = 0; i < deviceCount; i++) {
        printf("GPU%d     ", i);
        for (int j = 0; j < deviceCount; j++) {
            if (i == j) {
                printf("----     ");
            } else {
                int canAccessPeer;
                CUDA_CHECK(hipDeviceCanAccessPeer(&canAccessPeer, i, j));
                
                if (canAccessPeer) {
                    double bandwidth = measureP2PBandwidth(i, j, testSize);
                    printf("%.2f GB/s", bandwidth);
                } else {
                    printf("No P2P   ");
                }
            }
        }
        printf("\n");
    }
    printf("\n");
    
    // Demonstrate asynchronous P2P transfers
    if (deviceCount >= 2) {
        demonstrateAsyncP2P(0, 1);
        printf("\n");
    }
    
    // Ring communication pattern
    if (deviceCount >= 3) {
        demonstrateRingCommunication(deviceCount);
        printf("\n");
    }
    
    // All-reduce operation
    if (deviceCount >= 2) {
        demonstrateAllReduce(deviceCount);
        printf("\n");
    }
    
    // Performance analysis
    printf("=== Performance Analysis ===\n");
    printf("Key Observations:\n");
    printf("1. P2P bandwidth varies significantly between GPU pairs\n");
    printf("2. NVLink provides much higher bandwidth than PCIe\n");
    printf("3. Asynchronous transfers can overlap with computation\n");
    printf("4. Communication patterns affect overall performance\n");
    
    printf("\nBest Practices:\n");
    printf("- Check P2P capabilities before enabling access\n");
    printf("- Use asynchronous transfers when possible\n");
    printf("- Consider topology when designing communication patterns\n");
    printf("- Minimize data movement between GPUs\n");
    printf("- Use streams to overlap communication and computation\n");
    
    printf("\nP2P Communication demonstration completed!\n");
    return 0;
}